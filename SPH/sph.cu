#include "hip/hip_runtime.h"
#include "sph.cuh"
#include <math.h>
#include <stdio.h>



// Poly6 kernel
__device__ float poly6(float r2, float h) {
    float h2 = h * h;
    if (r2 > h2) return 0.0f;
    float term = h2 - r2;
    return (315.0f / (64.0f * 3.14159f * powf(h, 9))) * term * term * term;
}

// Spiky kernel gradient (softened)
__device__ float2 spikyGrad(float2 r, float r_len, float h) {
    if (r_len > h || r_len < 1e-6) return make_float2(0, 0);
    float term = h - r_len;
    float coeff = -45.0f / (3.14159f * powf(h, 6)) * term * term / r_len;
    return make_float2(coeff * r.x, coeff * r.y);
}

// Viscosity kernel Laplacian
__device__ float viscosityLaplacian(float r, float h) {
    if (r > h) return 0.0f;
    return 45.0f / (3.14159f * powf(h, 6)) * (h - r);
}

// Lennard-Jones force
__device__ float2 ljForce(float2 r, float r_len, float sigma, float epsilon) {
    if (r_len >= 2.5f * sigma || r_len < 1e-6f) return make_float2(0, 0);
    float inv_r = 1.0f / r_len;
    float sigma_over_r = sigma * inv_r;
    float sigma_over_r2 = sigma_over_r * sigma_over_r;
    float sigma_over_r6 = sigma_over_r2 * sigma_over_r2 * sigma_over_r2;
    float sigma_over_r12 = sigma_over_r6 * sigma_over_r6;
    float inv_r2 = inv_r * inv_r;
    float coeff = 24.0f * epsilon * (2.0f * sigma_over_r12 - sigma_over_r6) * inv_r2;
    return make_float2(coeff * r.x, coeff * r.y);
}

__global__ void computeDensityPressure(Particle* particles, int* grid, int* cellStart, int* cellEnd) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    Particle& p = particles[i];
    p.density = 0.0f;

    for (int j = 0; j < N; j++) {
        float2 r;
        r.x = p.pos.x - particles[j].pos.x;
        r.y = p.pos.y - particles[j].pos.y;
        float r2 = r.x * r.x + r.y * r.y;
        if (r2 < H * H) {
            p.density += MASS * poly6(r2, H);
        }
    }
    p.pressure = STIFFNESS * (p.density - REST_DENSITY);
    if (p.pressure < 0.0f) p.pressure = 0.0f;
}

__global__ void computeForces(Particle* particles, int* grid, int* cellStart, int* cellEnd) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    Particle& p = particles[i];
    float2 force = make_float2(0, 0);

    for (int j = 0; j < N; j++) {
        if (j == i)
            continue;
        float2 r;
        r.x = p.pos.x - particles[j].pos.x;
        r.y = p.pos.y - particles[j].pos.y;
        float r2 = r.x * r.x + r.y * r.y;
        if (r2 >= H * H) continue;
        float r_len = sqrtf(r2);

        // Pressure force
        float2 grad = spikyGrad(r, r_len, H);
        float pressureTerm = (p.pressure + particles[j].pressure) / (2.0f * particles[j].density);
        float2 pressureForce = make_float2(-MASS * pressureTerm * grad.x, -MASS * pressureTerm * grad.y);

        // Viscosity force
        float2 relVel;
        relVel.x = particles[j].vel.x - p.vel.x;
        relVel.y = particles[j].vel.y - p.vel.y;
        float viscForce = VISCOSITY * MASS * viscosityLaplacian(r_len, H) / particles[j].density;

        // Lennard-Jones force
        float2 ljF = ljForce(r, r_len, SIGMA, EPSILON);

        // Accumulate forces
        force.x += pressureForce.x + viscForce * relVel.x + ljF.x;
        force.y += pressureForce.y + viscForce * relVel.y + ljF.y;
    }
    // Gravity
    force.y -= 9.81f * p.density;

    // Cap force magnitude
    float forceMag = sqrtf(force.x * force.x + force.y * force.y);
    if (forceMag > 1000.0f * p.density) {
        float scale = 1000.0f * p.density / forceMag;
        force.x *= scale;
        force.y *= scale;
    }

    // Update velocity
    p.vel.x += DT * force.x / p.density;
    p.vel.y += DT * force.y / p.density;
}

__global__ void integrate(Particle* particles) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    Particle& p = particles[i];
    p.pos.x += DT * p.vel.x;
    p.pos.y += DT * p.vel.y;

    // Cap velocity
    float velMag = sqrtf(p.vel.x * p.vel.x + p.vel.y * p.vel.y);
    if (velMag > MAX_VEL) {
        float scale = MAX_VEL / velMag;
        p.vel.x *= scale;
        p.vel.y *= scale;
    }

    // Boundary conditions
    if (p.pos.x < 0.05f) {
        p.pos.x = 0.05f;
        p.vel.x = fabsf(p.vel.x) * DAMPING;
    }
    if (p.pos.x > 1.95f) {
        p.pos.x = 1.95f;
        p.vel.x = -fabsf(p.vel.x) * DAMPING;
    }
    if (p.pos.y < 0.05f) {
        p.pos.y = 0.05f;
        p.vel.y = fabsf(p.vel.y) * DAMPING;
    }
    if (p.pos.y > 1.95f) {
        p.pos.y = 1.95f;
        p.vel.y = -fabsf(p.vel.y) * DAMPING;
    }
}

void initSimulation(Particle* particles, hipGraphicsResource* cudaVBO) {
    Particle* d_particles;
    CUDA_CHECK(hipMalloc(&d_particles, N * sizeof(Particle)));
    Particle h_particles[N];
    for (int i = 0; i < N; i++) {
        h_particles[i].pos = make_float2(0.7f + 0.05f * (i % 20), 1.5f - 0.05f * (i / 20));
        h_particles[i].vel = make_float2(0, 0);
        h_particles[i].density = REST_DENSITY;
        h_particles[i].pressure = 0;
    }
    CUDA_CHECK(hipMemcpy(d_particles, h_particles, N * sizeof(Particle), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(particles, d_particles, N * sizeof(Particle), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_particles));
}

void stepSimulation(Particle* particles, int* grid, int* cellStart, int* cellEnd, hipGraphicsResource* cudaVBO) {
    Particle* d_particles;
    CUDA_CHECK(hipMalloc(&d_particles, N * sizeof(Particle)));
    CUDA_CHECK(hipMemcpy(d_particles, particles, N * sizeof(Particle), hipMemcpyHostToDevice));

    int threadsPerBlock = 256;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    computeDensityPressure << <blocks, threadsPerBlock >> > (d_particles, nullptr, nullptr, nullptr);
    CUDA_CHECK(hipDeviceSynchronize());
    computeForces << <blocks, threadsPerBlock >> > (d_particles, nullptr, nullptr, nullptr);
    CUDA_CHECK(hipDeviceSynchronize());
    integrate << <blocks, threadsPerBlock >> > (d_particles);
    CUDA_CHECK(hipDeviceSynchronize());

    float2* vboPtr;
    size_t size;
    CUDA_CHECK(hipGraphicsMapResources(1, &cudaVBO, 0));
    CUDA_CHECK(hipGraphicsResourceGetMappedPointer((void**)&vboPtr, &size, cudaVBO));
    CUDA_CHECK(hipMemcpy(vboPtr, d_particles, N * sizeof(float2), hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipGraphicsUnmapResources(1, &cudaVBO, 0));

    CUDA_CHECK(hipMemcpy(particles, d_particles, N * sizeof(Particle), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_particles));
}